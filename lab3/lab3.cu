#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  hipdnnStatus_t err = (f); \
  if (err != HIPDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

//const int H = 1024, W = 1024;
const int H = 1024, W = 1024;
const int C = 3, FW = 3, FH = 3, K = 64;
const int P = 1;
const int H0 = H + 2 * P;
const int W0 = W + 2 * P;
const int INPUT_SIZE = C * H * W;
const int FILTER_SIZE = C * K * FW * FH;
const int OUTPUT_SIZE = K * H * W;
const int INPUT_PADDED_SIZE = C * H0 * W0;

struct Configuration {
    int K, C, H, W, P, H0, W0, FW, FH;
};

template<class T>
T &at(T *tensor, int c, int i, int j, int height, int width) {
    return tensor[c * height * width + i * width + j];
}

template<class T>
T &at(T *tensor, int k, int c, int i, int j, int layer, int height, int width) {
    return tensor[k * layer * height * width + c * height * width + i * width + j];
}

__device__
inline double &at_d(double *tensor, int c, int i, int j, int height, int width) {
    return tensor[c * height * width + i * width + j];
}

__device__
inline double &at_d(double *tensor, int k, int c, int i, int j, int layer, int height, int width) {
    return tensor[k * layer * height * width + c * height * width + i * width + j];
}

inline int ceil(int a, int b) {
    return (a+b-1)/b;
}

//////////////////////////////////////////////////////////////////////
void init_input(double *input) {
    // real input
    for (int c = 0; c < C; c++) {
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                at(input, c, x, y, H, W) = c * (x + y);
            }
        }
    }
}

void clear_output(double *output) {
    std::fill(output, output+OUTPUT_SIZE, 0);
}

void add_padding(double *raw, double *padded) {
    // padding, p=1, set to zero
    for (int c = 0; c < C; c++) {
        // top and bottom
        for (int j = 0; j < W0; j++) {
            at(padded, c, 0, j, H0, W0) = 0;
            at(padded, c, H0 - 1, j, H0, W0) = 0;
        }
        // left and right
        for (int i = 0; i < H0; i++) {
            at(padded, c, i, 0, H0, W0) = 0;
            at(padded, c, i, W0 - 1, H0, W0) = 0;
        }
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                at(padded, c, x+1, y+1, H0, W0) = at(raw, c, x, y, H, W);
            }
        }
    }
}

void init_filter(double *filter) {
    for (int k = 0; k < K; k++) {
        for (int c = 0; c < C; c++) {
            for (int i = 0; i < FH; i++) {
                for (int j = 0; j < FW; j++) {
                    at(filter, k, c, i, j, C, FH, FW) = (c + k) * (i + j);
                }
            }
        }
    }
}

void print_mat(double *mat, int channel, int height, int width) {
    for (int c = 0; c < channel; c++) {
        for (int i = 0 ; i < height; i++) {
            for (int j = 0 ; j < width; j++) {
                std::cout << at(mat, c, i, j, height, width) << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
    std::cout << std::endl;
}

double calc_checksum(double *tensor, int layer, int height, int width) {
    double sum = 0;
    for (int c = 0; c < layer; c++) {
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                sum += at(tensor, c, i, j, height, width);
            }
        }
    }
    return sum;
}


//////////////////////////////////////////////////////////////////////
void naive_convolution(double *input, double *filter, double *output) {
    for (int k = 0; k < K; k++) {
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                // conv sum
                double sum = 0;
                for (int c = 0; c < C; c++) {
                    for (int j = 0; j < FH; j++) {
                        for (int i = 0; i < FW; i++) {
                            sum += at(filter, k, c, FW - 1 - i, FH - 1 - j, C, FW, FH) *
                                   at(input, c, x + i, y + j, H0, W0);
                        }
                    }
                }
                at(output, k, x, y, H, W) = sum;
            }
        }
    }
}

//////////////////////////////////////////////////////

__global__ void naive_cuda_kernel(double *input, double *filter, double *output,
                                  int K_d, int C_d, int H_d, int W_d, int H0_d, int W0_d, int FW_d, int FH_d) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
    double sum = 0;

    if (k < K_d && x < H0_d && y < W0_d) {
        for (int c = 0; c < C_d; c++) {
            for (int j = 0; j < FH_d; j++) {
                for (int i = 0; i < FW_d; i++) {
                    sum += at_d(filter, k, c, FW_d - 1 - i, FH_d - 1 - j, C_d, FW_d, FH_d) *
                           at_d(input, c, x + i, y + j, H0_d, W0_d);
                }
            }
        }
        at_d(output, k, x, y, H_d, W_d) = sum;
    }
}

void run_naive_cuda(double *input, double *filter, double *output) {
    double *input_d, *filter_d, *output_d;
    CUDA_CALL(hipMalloc(&input_d, INPUT_SIZE * sizeof(double)));
    CUDA_CALL(hipMalloc(&filter_d, FILTER_SIZE * sizeof(double)));
    CUDA_CALL(hipMalloc(&output_d, OUTPUT_SIZE * sizeof(double)));
    CUDA_CALL(hipMemcpy(input_d, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(filter_d, filter, FILTER_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(output_d, output, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    int TILE_LEN = 512;
    int CHAN_LEN = 16;
    dim3 grid(ceil(H0, TILE_LEN), ceil(W0, TILE_LEN), ceil(K, CHAN_LEN));
    dim3 block(TILE_LEN, TILE_LEN, CHAN_LEN);
    CUDA_CALL(naive_cuda_kernel<<< grid, block >>>(input_d, filter_d, output_d, K, C, H, W, H0, W0, FW, FH));
    hipDeviceSynchronize();

    // copy back
    CUDA_CALL(hipMemcpy(output, output_d, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    // finalizing
    CUDA_CALL(hipFree(output_d));
    CUDA_CALL(hipFree(filter_d));
    CUDA_CALL(hipFree(input_d));

}

//////////////////////////////////////////////////////
void run_cudnn(double *input, double *filter, double *output) {

    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));

    // define input descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    // desc, format, data type, channels, height, width
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));
    // copy data from host to device
    double *input_d;
    CUDA_CALL(hipMalloc(&input_d, INPUT_SIZE * sizeof(double)));
    CUDA_CALL(hipMemcpy(input_d, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    // define filter descriptor
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW));
    double *filter_d;
    CUDA_CALL(hipMalloc(&filter_d, FILTER_SIZE * sizeof(double)));
    CUDA_CALL(hipMemcpy(filter_d, filter, FILTER_SIZE * sizeof(double), hipMemcpyHostToDevice));

    // define output descriptor
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));
    double *output_d;
    CUDA_CALL(hipMalloc(&output_d, OUTPUT_SIZE* sizeof(double)));
    CUDA_CALL(hipMemcpy(output_d, output, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    // define conv pre action
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    // desc, pad h, pad w, vertical stride, horizontal stride, dilation height, dilation width, mode, data type
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor, P, P, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

    // define algo
    hipdnnConvolutionFwdAlgo_t algorithm;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, filter_descriptor, conv_descriptor, output_descriptor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algorithm));

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, filter_descriptor, conv_descriptor, output_descriptor, algorithm, &ws_size));
    double *ws_data;
    CUDA_CALL(hipMalloc(&ws_data, ws_size));


    // perform conv !!!!!!!!1
    double alpha = 1. , beta = 0.;
    CUDNN_CALL(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, input_d, filter_descriptor, filter_d,
                                       conv_descriptor, algorithm, ws_data, ws_size, &beta, output_descriptor, output_d));

    // copy back
    CUDA_CALL(hipMemcpy(output, output_d, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    // finalizing
    CUDA_CALL(hipFree(ws_data));
    CUDA_CALL(hipFree(output_d));
    CUDA_CALL(hipFree(filter_d));
    CUDA_CALL(hipFree(input_d));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroy(cudnn));

}

int main() {

    // malloc
    auto *input = (double *) malloc(sizeof(double) * INPUT_SIZE);
    auto *input_padded = (double *) malloc(sizeof(double) * INPUT_PADDED_SIZE);
    auto *filter = (double *) malloc(sizeof(double) * FILTER_SIZE);
    auto *output = (double *) malloc(sizeof(double) * OUTPUT_SIZE);
    // init zero
    std::fill(input, input + INPUT_SIZE, 0);
    std::fill(input_padded, input_padded + INPUT_PADDED_SIZE, 0);
    std::fill(filter, filter + FILTER_SIZE, 0);
    std::fill(output, output + OUTPUT_SIZE, 0);

    // init
    init_input(input);
    add_padding(input, input_padded);
    init_filter(filter);

    double checksum;

    // naive conv cpu mode
//    naive_convolution(input_padded, filter, output);
//    checksum = calc_checksum(output, K, H, W);
//    std::cout << checksum << std::endl;

    // cuda
    clear_output(output);
    run_naive_cuda(input, filter, output);
    checksum = calc_checksum(output, K, H, W);
    std::cout << checksum << std::endl;

    // cuda tiled

    // cuDNN
    clear_output(output);
    run_cudnn(input, filter, output);
    checksum = calc_checksum(output, K, H, W);
    std::cout << checksum << std::endl;

    return 0;
}
